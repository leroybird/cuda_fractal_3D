#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <SFML/Graphics.hpp>

#include <SFML/Graphics.hpp>
#include <algorithm>
#include <stdint.h>
#include "math.h"
#include "simple_math.h"
#include <stdio.h>

const int WIDTH = 1024;
const int HEIGHT = 1024;
const int IMG_CH = 4;

// Maximum iterations for calculating the mandelbulb iterations
const int MAX_ITER = 200;
// Stop marching when we get too close
const float MIN_DIST = 1e-5;

// How many ray marches we before stopping
const int MAX_RAY_ITER = 128;


void mandelbrotGPU(sf::Uint8 *, float);

#define cudaAssertSuccess(ans)                     \
  {                                                \
    _cudaAssertSuccess((ans), __FILE__, __LINE__); \
  }
inline void _cudaAssertSuccess(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "_cudaAssertSuccess: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

__host__ __device__ float boxFold(float component)
{
  if (component > 1)
    component = 2 - component;
  else if (component < -1)
    component = -2 - component;
  return component;
}

// Define the distance estimation function so we can dynamically change the distance function at runtime
typedef float (*distanceFunction)(float3, float);

__host__ __device__ float distEstMandelBox(float3 pos, float time)
{
  // Animation Code
  // For MandelBox, we aviod the range [1, -1] and instead display from [-3, -1] and [1, 3]
  float scale = fmodf(time / 4, 4) - 2;
  scale =  scale < 0 ? scale - 1 : scale + 1;

  // Returns the distance (magnitude) from a point to the mandelbulb fractal.
  float3 offset = pos;
  float dr = 1.0;

  for (int n = 0; n < MAX_ITER; n++)
  {
    // bail out
    if (length(pos) > 16)
      break;

    // box fold
    pos = make_float3(boxFold(pos.x), boxFold(pos.y), boxFold(pos.z));

    // Sphere fold
    float mag = length(pos);
    if (mag < 0.5)
    {
      pos = pos * 4;
    }
    else if (mag < 1)
    {
      pos = pos * (1.0 / (mag * mag));
    }

    pos = pos * scale + offset;
    dr = dr * abs(scale) + 1.0;
  }

  return length(pos) / abs(dr);
}

__host__ __device__ float distEstMandelBulb(float3 pos, float time)
{
  // Returns the distance (magnitude) from a point to the mandelbulb fractal. Based off
  // http://blog.hvidtfeldts.net/index.php/2011/09/distance-estimated-3d-fractals-v-the-mandelbulb-different-de-approximations/
  float3 z = pos;

  // Animation code
  float power = fmodf(time / 3, 3) + 1;
  power =  power * power;
  
  float dr = 1.0;
  float r = 0.0;
  for (int i = 0; i < MAX_ITER; i++)
  {
    //bail out.
    r = length(z);
    if (r > 16)
      break;

    // Convert to polar coords
    float theta = acosf(z.z / r);
    float phi = atan2f(z.y, z.x);
    dr = powf(r, power - 1.0) * power * dr + 1.0;

    float zr = powf(r, power);
    theta = theta * power;
    phi = phi * power;

    // Back to euclidean
    z = make_float3(sin(theta) * cos(phi), sin(phi) * sin(theta), cos(theta)) * zr;
    z = z + pos;
  }

  return 0.5 * log(r) * r / dr;
}

__host__ __device__ float march(float3 origin, float3 direction, float time, distanceFunction func)
{
  // We can slowly march foward in the current direction upto the maximum number of ray iterations.
  // The distance esimation fractal is passed as an argument.
  int steps = 0;
  float total_dist = 0;
  for (; steps < MAX_RAY_ITER; steps++)
  {
    float3 p = origin + direction * total_dist;
    float distance = func(p, time);
    total_dist += distance;
    if (distance < MIN_DIST)
      break;
  }

  return 1.0 - (float)steps / (float)MAX_RAY_ITER;
}


__global__ void calculateBuffer(uint8_t *image_buffer, float power, float3 rayOrigin,  distanceFunction func)
{
  // TODO: dynamically change the distance estimation function
  //distanceFunction func = distEstMandelBulb;

  int row = blockIdx.y * blockDim.y + threadIdx.y; // WIDTH
  int col = blockIdx.x * blockDim.x + threadIdx.x; // HEIGHT
  int idx = IMG_CH * (row * WIDTH + col);
  if (col >= WIDTH || row >= HEIGHT)
    return;


  // Calculate the ray origin from the centre of the pixel
  float x0 = ((float)col / WIDTH) * 2.0f - 1.0f;
  float y0 = ((float)row / HEIGHT) * 2.0f - 1.0f;
  float3 center_dir = norm(make_float3(0, 0, 0) - rayOrigin);
  float3 xDir = norm(cross(center_dir, make_float3(0, 1, 0)));
  float3 yDir = norm(cross(center_dir, xDir));

  float3 pixelOrigin = rayOrigin + xDir * x0 + yDir * y0 + center_dir;
  float3 direction = pixelOrigin - rayOrigin;

  float colour = march(pixelOrigin, direction, power, func);

  // Convert the distance into a colour, for now just go with purple as it looks good.
  image_buffer[idx] = (uint8_t)255 * (0.5 * colour);
  image_buffer[idx + 1] = (uint8_t)0;
  image_buffer[idx + 2] = (uint8_t)255 * colour;
  image_buffer[idx + 3] = 255;
}

void runKernel(sf::Uint8 *image_buffer, float power, float3 rayOrigin, distanceFunction func)
{
  // Runs the CUDA kernel and copies the result back to memory.
  uint8_t *d_image_buffer;
  cudaAssertSuccess(hipMalloc(&d_image_buffer, WIDTH * HEIGHT * IMG_CH));
  dim3 block_size(16, 16);
  dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);
  calculateBuffer<<<grid_size, block_size>>>(d_image_buffer, power, rayOrigin, func);

  cudaAssertSuccess(hipPeekAtLastError());
  cudaAssertSuccess(hipDeviceSynchronize());
  cudaAssertSuccess(hipMemcpy(image_buffer, d_image_buffer, IMG_CH * HEIGHT * WIDTH, hipMemcpyDeviceToHost));
  cudaAssertSuccess(hipFree(d_image_buffer));
}

__device__ distanceFunction p_bulbDev = distEstMandelBulb;
__device__ distanceFunction p_boxDev = distEstMandelBox;


int renderLoop()
{
  sf::RenderWindow window(sf::VideoMode(HEIGHT, WIDTH), "3D fractal viewer");

  sf::Texture text;
  text.create(HEIGHT, WIDTH);
  sf::Sprite imgSprite(text);

  int buf_size = WIDTH * HEIGHT * IMG_CH;
  sf::Uint8 *pixels = new sf::Uint8[buf_size];

  sf::Clock clock;
  clock.restart();

  bool changePower = true;
  bool isMandel = true;
  float3 rayOrigin = make_float3(0.f, 0.f, -2.f);

  float draw_time = 0;

  double hAngle = 0;
  double vAngle = 0;

  double viewR = 1.0;

  float renderTime = 0.0;

  // Setup distanace estimation pointers
  distanceFunction boxDistance;
  distanceFunction bulbDistance;
  distanceFunction * currDistFunc =  &boxDistance;
	hipMemcpyFromSymbol(&boxDistance, HIP_SYMBOL(p_boxDev), sizeof(distanceFunction));
	hipMemcpyFromSymbol(&bulbDistance, HIP_SYMBOL(p_bulbDev), sizeof(distanceFunction));

  //
  while (window.isOpen())
  {
    sf::Event event;
    while (window.pollEvent(event))
    {
      if (event.type == sf::Event::Closed)
        window.close();

      if (event.type == sf::Event::MouseWheelScrolled)
      {
        if (event.mouseWheelScroll.wheel == sf::Mouse::VerticalWheel)
        {
          float delta = event.mouseWheelScroll.delta * 0.1;
          viewR -= delta;
          if (viewR < 0)
          {
            viewR = 0;
          }
        }
      }
      else if (event.type == sf::Event::KeyPressed)
      {
        if (event.key.code == sf::Keyboard::P)
        {
          changePower = !changePower;
        }
        else if (event.key.code == sf::Keyboard::M)
        {
          isMandel = !isMandel;
        }
      }
    }

    // Use the draw time to control the amount of seconds to spend while
    draw_time = clock.restart().asSeconds();


    // Control the rotation
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Right))
    {
      hAngle += 1.0 * draw_time;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Left))
    {
      hAngle -= 1.0 * draw_time;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Up))
    {
      vAngle += 1.0 * draw_time;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Down))
    {
      vAngle -= 1.0 * draw_time;
    }


    // TODO: refactor this into a list as more fractal types are added.
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Num1))
    {
      currDistFunc = &boxDistance;
    }
    if (sf::Keyboard::isKeyPressed(sf::Keyboard::Num2))
    {
      currDistFunc = &bulbDistance;
    }

    rayOrigin.x = sin(hAngle) * cos(vAngle);
    rayOrigin.y = sin(vAngle) * sin(hAngle);
    rayOrigin.z = cos(hAngle);
    rayOrigin = rayOrigin * viewR;

    if (changePower)
    {
      // Slowly change the power
      renderTime += draw_time;
    }

    // Currently transfering from the GPU back to the CPU,
    window.clear();
    runKernel(pixels, renderTime, rayOrigin, *currDistFunc);
    text.update(pixels);
    window.draw(imgSprite);
    window.display();

    std::cout << vAngle << " " << hAngle << " " << std::endl;
    std::cout << rayOrigin.x << " " << rayOrigin.y << " " << rayOrigin.z << std::endl;
    float fps = 1.f / draw_time;
    std::cout << "fps: " << fps << '\n';
  }

  return 0;
}

int main(int argc, char **argv)
{
  return renderLoop();
}